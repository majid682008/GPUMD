#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the rigid-ion potential.

Reference for the method of evaluating the Coulomb force in the rigid-ion
potential:

[1] C. J. Fennell and J. D. Gezelter. Is the Ewald summation still necessary?
Pairwise alternatives to the accepted standard for long-range electrostatics,
J. Chem. Phys. 124, 234104 (2006).
------------------------------------------------------------------------------*/

#include "ri.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"

#define BLOCK_SIZE_FORCE 128
#define RI_ALPHA 0.2
#define RI_ALPHA_SQ 0.04
#define RI_PI_FACTOR 0.225675833419103 // ALPHA * 2 / SQRT(PI)

RI::RI(FILE* fid, int num_types)
{
  printf("Use the rigid-ion potential.\n");

  int N = num_types*(num_types+1)/2;

  fscanf(fid, "%lf", &ri_para.cutoff);

  double q[num_types];
  for (int i = 0; i < num_types; i++) {
    fscanf(fid, "%lf", &q[i]);
  }

  double x[N][3];
  for (int i = 0; i < N; i++) {
    int count = fscanf(fid, "%lf%lf%lf", &x[i][0], &x[i][1], &x[i][2]);
    PRINT_SCANF_ERROR(count, 3, "Reading error for rigid-ion potential.");
  }

  if (num_types == 2){
    ri_para.a11 = x[0][0];
    ri_para.b11 = 1.0 / x[0][1];
    ri_para.c11 = x[0][2];
    ri_para.a12 = x[1][0];
    ri_para.b12 = 1.0 / x[1][1];
    ri_para.c12 = x[1][2];
    ri_para.a22 = x[2][0];
    ri_para.b22 = 1.0 / x[2][1];
    ri_para.c22 = x[2][2];
    ri_para.qq11 = q[0] * q[0] * K_C;
    ri_para.qq12 = q[0] * q[1] * K_C;
    ri_para.qq22 = q[1] * q[1] * K_C;
  }
  else if (num_types == 3){
    ri_para.a11 = x[0][0];
    ri_para.b11 = 1.0 / x[0][1];
    ri_para.c11 = x[0][2];
    ri_para.a12 = x[1][0];
    ri_para.b12 = 1.0 / x[1][1];
    ri_para.c12 = x[1][2];
    ri_para.a13 = x[2][0];
    ri_para.b13 = 1.0 / x[2][1];
    ri_para.c13 = x[2][2];
    ri_para.a22 = x[3][0];
    ri_para.b22 = 1.0 / x[3][1];
    ri_para.c22 = x[3][2];
    ri_para.a23 = x[4][0];
    ri_para.b23 = 1.0 / x[4][1];
    ri_para.c23 = x[4][2];
    ri_para.a33 = x[5][0];
    ri_para.b33 = 1.0 / x[5][1];
    ri_para.c33 = x[5][2];
    ri_para.qq11 = q[0] * q[0] * K_C;
    ri_para.qq12 = q[0] * q[1] * K_C;
    ri_para.qq13 = q[0] * q[2] * K_C;
    ri_para.qq22 = q[1] * q[1] * K_C;
    ri_para.qq23 = q[1] * q[2] * K_C;
    ri_para.qq33 = q[2] * q[2] * K_C;
  }
  else if (num_types == 4){
    ri_para.a11 = x[0][0];
    ri_para.b11 = 1.0 / x[0][1];
    ri_para.c11 = x[0][2];
    ri_para.a12 = x[1][0];
    ri_para.b12 = 1.0 / x[1][1];
    ri_para.c12 = x[1][2];
    ri_para.a13 = x[2][0];
    ri_para.b13 = 1.0 / x[2][1];
    ri_para.c13 = x[2][2];
    ri_para.a14 = x[3][0];
    ri_para.b14 = 1.0 / x[3][1];
    ri_para.c14 = x[3][2];
    ri_para.a22 = x[4][0];
    ri_para.b22 = 1.0 / x[4][1];
    ri_para.c22 = x[4][2];
    ri_para.a23 = x[5][0];
    ri_para.b23 = 1.0 / x[5][1];
    ri_para.c23 = x[5][2];
    ri_para.a24 = x[6][0];
    ri_para.b24 = 1.0 / x[6][1];
    ri_para.c24 = x[6][2];
    ri_para.a33 = x[7][0];
    ri_para.b33 = 1.0 / x[7][1];
    ri_para.c33 = x[7][2];
    ri_para.a34 = x[8][0];
    ri_para.b34 = 1.0 / x[8][1];
    ri_para.c34 = x[8][2];
    ri_para.a44 = x[9][0];
    ri_para.b44 = 1.0 / x[9][1];
    ri_para.c44 = x[9][2];
    ri_para.qq11 = q[0] * q[0] * K_C;
    ri_para.qq12 = q[0] * q[1] * K_C;
    ri_para.qq13 = q[0] * q[2] * K_C;
    ri_para.qq14 = q[0] * q[3] * K_C;
    ri_para.qq22 = q[1] * q[1] * K_C;
    ri_para.qq23 = q[1] * q[2] * K_C;
    ri_para.qq24 = q[1] * q[3] * K_C;
    ri_para.qq33 = q[2] * q[2] * K_C;
    ri_para.qq34 = q[2] * q[3] * K_C;
    ri_para.qq44 = q[3] * q[3] * K_C;
  }
  else if (num_types == 5){
    ri_para.a11 = x[0][0];
    ri_para.b11 = 1.0 / x[0][1];
    ri_para.c11 = x[0][2];
    ri_para.a12 = x[1][0];
    ri_para.b12 = 1.0 / x[1][1];
    ri_para.c12 = x[1][2];
    ri_para.a13 = x[2][0];
    ri_para.b13 = 1.0 / x[2][1];
    ri_para.c13 = x[2][2];
    ri_para.a14 = x[3][0];
    ri_para.b14 = 1.0 / x[3][1];
    ri_para.c14 = x[3][2];
    ri_para.a15 = x[4][0];
    ri_para.b15 = 1.0 / x[4][1];
    ri_para.c15 = x[4][2];
    ri_para.a22 = x[5][0];
    ri_para.b22 = 1.0 / x[5][1];
    ri_para.c22 = x[5][2];
    ri_para.a23 = x[6][0];
    ri_para.b23 = 1.0 / x[6][1];
    ri_para.c23 = x[6][2];
    ri_para.a24 = x[7][0];
    ri_para.b24 = 1.0 / x[7][1];
    ri_para.c24 = x[7][2];
    ri_para.a25 = x[8][0];
    ri_para.b25 = 1.0 / x[8][1];
    ri_para.c25 = x[8][2];
    ri_para.a33 = x[9][0];
    ri_para.b33 = 1.0 / x[9][1];
    ri_para.c33 = x[9][2];
    ri_para.a34 = x[10][0];
    ri_para.b34 = 1.0 / x[10][1];
    ri_para.c34 = x[10][2];
    ri_para.a35 = x[11][0];
    ri_para.b35 = 1.0 / x[11][1];
    ri_para.c35 = x[11][2];
    ri_para.a44 = x[12][0];
    ri_para.b44 = 1.0 / x[12][1];
    ri_para.c44 = x[12][2];
    ri_para.a45 = x[13][0];
    ri_para.b45 = 1.0 / x[13][1];
    ri_para.c45 = x[13][2];
    ri_para.a55 = x[14][0];
    ri_para.b55 = 1.0 / x[14][1];
    ri_para.c55 = x[14][2];
    ri_para.qq11 = q[0] * q[0] * K_C;
    ri_para.qq12 = q[0] * q[1] * K_C;
    ri_para.qq13 = q[0] * q[2] * K_C;
    ri_para.qq14 = q[0] * q[3] * K_C;
    ri_para.qq15 = q[0] * q[4] * K_C;
    ri_para.qq22 = q[1] * q[1] * K_C;
    ri_para.qq23 = q[1] * q[2] * K_C;
    ri_para.qq24 = q[1] * q[3] * K_C;
    ri_para.qq25 = q[1] * q[4] * K_C;
    ri_para.qq33 = q[2] * q[2] * K_C;
    ri_para.qq34 = q[2] * q[3] * K_C;
    ri_para.qq35 = q[2] * q[4] * K_C;
    ri_para.qq44 = q[3] * q[3] * K_C;
    ri_para.qq45 = q[3] * q[4] * K_C;
    ri_para.qq55 = q[4] * q[4] * K_C;
  }
  else if (num_types == 6){
    ri_para.a11 = x[0][0];
    ri_para.b11 = 1.0 / x[0][1];
    ri_para.c11 = x[0][2];
    ri_para.a12 = x[1][0];
    ri_para.b12 = 1.0 / x[1][1];
    ri_para.c12 = x[1][2];
    ri_para.a13 = x[2][0];
    ri_para.b13 = 1.0 / x[2][1];
    ri_para.c13 = x[2][2];
    ri_para.a14 = x[3][0];
    ri_para.b14 = 1.0 / x[3][1];
    ri_para.c14 = x[3][2];
    ri_para.a15 = x[4][0];
    ri_para.b15 = 1.0 / x[4][1];
    ri_para.c15 = x[4][2];
    ri_para.a16 = x[5][0];
    ri_para.b16 = 1.0 / x[5][1];
    ri_para.c16 = x[5][2];
    ri_para.a22 = x[6][0];
    ri_para.b22 = 1.0 / x[6][1];
    ri_para.c22 = x[6][2];
    ri_para.a23 = x[7][0];
    ri_para.b23 = 1.0 / x[7][1];
    ri_para.c23 = x[7][2];
    ri_para.a24 = x[8][0];
    ri_para.b24 = 1.0 / x[8][1];
    ri_para.c24 = x[8][2];
    ri_para.a25 = x[9][0];
    ri_para.b25 = 1.0 / x[9][1];
    ri_para.c25 = x[9][2];
    ri_para.a26 = x[10][0];
    ri_para.b26 = 1.0 / x[10][1];
    ri_para.c26 = x[10][2];
    ri_para.a33 = x[11][0];
    ri_para.b33 = 1.0 / x[11][1];
    ri_para.c33 = x[11][2];
    ri_para.a34 = x[12][0];
    ri_para.b34 = 1.0 / x[12][1];
    ri_para.c34 = x[12][2];
    ri_para.a35 = x[13][0];
    ri_para.b35 = 1.0 / x[13][1];
    ri_para.c35 = x[13][2];
    ri_para.a36 = x[14][0];
    ri_para.b36 = 1.0 / x[14][1];
    ri_para.c36 = x[14][2];
    ri_para.a44 = x[15][0];
    ri_para.b44 = 1.0 / x[15][1];
    ri_para.c44 = x[15][2];
    ri_para.a45 = x[16][0];
    ri_para.b45 = 1.0 / x[16][1];
    ri_para.c45 = x[16][2];
    ri_para.a46 = x[17][0];
    ri_para.b46 = 1.0 / x[17][1];
    ri_para.c46 = x[17][2];
    ri_para.a55 = x[18][0];
    ri_para.b55 = 1.0 / x[18][1];
    ri_para.c55 = x[18][2];
    ri_para.a56 = x[19][0];
    ri_para.b56 = 1.0 / x[19][1];
    ri_para.c56 = x[19][2];
    ri_para.a66 = x[20][0];
    ri_para.b66 = 1.0 / x[20][1];
    ri_para.c66 = x[20][2];
    ri_para.qq11 = q[0] * q[0] * K_C;
    ri_para.qq12 = q[0] * q[1] * K_C;
    ri_para.qq13 = q[0] * q[2] * K_C;
    ri_para.qq14 = q[0] * q[3] * K_C;
    ri_para.qq15 = q[0] * q[4] * K_C;
    ri_para.qq16 = q[0] * q[5] * K_C;
    ri_para.qq22 = q[1] * q[1] * K_C;
    ri_para.qq23 = q[1] * q[2] * K_C;
    ri_para.qq24 = q[1] * q[3] * K_C;
    ri_para.qq25 = q[1] * q[4] * K_C;
    ri_para.qq26 = q[1] * q[5] * K_C;
    ri_para.qq33 = q[2] * q[2] * K_C;
    ri_para.qq34 = q[2] * q[3] * K_C;
    ri_para.qq35 = q[2] * q[4] * K_C;
    ri_para.qq36 = q[2] * q[5] * K_C;
    ri_para.qq44 = q[3] * q[3] * K_C;
    ri_para.qq45 = q[3] * q[4] * K_C;
    ri_para.qq46 = q[3] * q[5] * K_C;
    ri_para.qq55 = q[4] * q[4] * K_C;
    ri_para.qq56 = q[4] * q[5] * K_C;
    ri_para.qq55 = q[5] * q[5] * K_C;
  }

  rc = ri_para.cutoff; // force cutoff

  ri_para.v_rc = erfc(RI_ALPHA * rc) / rc;
  ri_para.dv_rc = -erfc(RI_ALPHA * rc) / (rc * rc);
  ri_para.dv_rc -= RI_PI_FACTOR * exp(-RI_ALPHA_SQ * rc * rc) / rc;
}

RI::~RI(void)
{
  // nothing
}

// get U_ij and (d U_ij / d r_ij) / r_ij (the RI potential)
static __device__ void
find_p2_and_f2(int type1, int type2, RI_Para ri, double d12sq, double& p2, double& f2)
{
  double a, b, c, qq;

  if (type1 == 0 && type2 == 0) {
    a = ri.a11;
    b = ri.b11;
    c = ri.c11;
    qq = ri.qq11;
  } else if  ((type1 == 0 && type2 == 1) || (type1 == 1 && type2 == 0))  {
    a = ri.a12;
    b = ri.b12;
    c = ri.c12;
    qq = ri.qq12;
  } else if (type1 == 1 && type2 == 1) {
    a = ri.a22;
    b = ri.b22;
    c = ri.c22;
    qq = ri.qq22;
  } else if  ((type1 == 0 && type2 == 2) || (type1 == 2 && type2 == 0))  {
    a = ri.a13;
    b = ri.b13;
    c = ri.c13;
    qq = ri.qq13;
  } else if ((type1 == 1 && type2 == 2) || (type1 == 2 && type2 == 1))  {
    a = ri.a23;
    b = ri.b23;
    c = ri.c23;
    qq = ri.qq23;
  } else if (type1 == 2 && type2 == 2) {
    a = ri.a33;
    b = ri.b33;
    c = ri.c33;
    qq = ri.qq33;
  } else if  ((type1 == 0 && type2 == 3) || (type1 == 3 && type2 == 0))  {
    a = ri.a14;
    b = ri.b14;
    c = ri.c14;
    qq = ri.qq14;
  } else if ((type1 == 1 && type2 == 3) || (type1 == 3 && type2 == 1))  {
    a = ri.a24;
    b = ri.b24;
    c = ri.c24;
    qq = ri.qq24;
  } else if ((type1 == 2 && type2 == 3) || (type1 == 3 && type2 == 2))  {
    a = ri.a34;
    b = ri.b34;
    c = ri.c34;
    qq = ri.qq34;
  } else if (type1 == 3 && type2 == 3) {
    a = ri.a44;
    b = ri.b44;
    c = ri.c44;
    qq = ri.qq44;
  } else if  ((type1 == 0 && type2 == 4) || (type1 == 4 && type2 == 0))  {
    a = ri.a15;
    b = ri.b15;
    c = ri.c15;
    qq = ri.qq15;
  } else if ((type1 == 1 && type2 == 4) || (type1 == 4 && type2 == 1))  {
    a = ri.a25;
    b = ri.b25;
    c = ri.c25;
    qq = ri.qq25;
  } else if ((type1 == 2 && type2 == 4) || (type1 == 4 && type2 == 2))  {
    a = ri.a35;
    b = ri.b35;
    c = ri.c35;
    qq = ri.qq35;
  } else if ((type1 == 3 && type2 == 4) || (type1 == 4 && type2 == 3))  {
    a = ri.a45;
    b = ri.b45;
    c = ri.c45;
    qq = ri.qq45;
  } else if (type1 == 4 && type2 == 4) {
    a = ri.a55;
    b = ri.b55;
    c = ri.c55;
    qq = ri.qq55;
  } else if  ((type1 == 0 && type2 == 5) || (type1 == 5 && type2 == 0))  {
    a = ri.a16;
    b = ri.b16;
    c = ri.c16;
    qq = ri.qq16;
  } else if ((type1 == 1 && type2 == 5) || (type1 == 5 && type2 == 1))  {
    a = ri.a26;
    b = ri.b26;
    c = ri.c26;
    qq = ri.qq26;
  } else if ((type1 == 2 && type2 == 5) || (type1 == 5 && type2 == 2))  {
    a = ri.a36;
    b = ri.b36;
    c = ri.c36;
    qq = ri.qq36;
  } else if ((type1 == 3 && type2 == 5) || (type1 == 5 && type2 == 3))  {
    a = ri.a46;
    b = ri.b46;
    c = ri.c46;
    qq = ri.qq46;
  } else if ((type1 == 4 && type2 == 5) || (type1 == 5 && type2 == 4))  {
    a = ri.a56;
    b = ri.b56;
    c = ri.c56;
    qq = ri.qq56;
  } else if (type1 == 5 && type2 == 5) {
    a = ri.a66;
    b = ri.b66;
    c = ri.c66;
    qq = ri.qq66;
  }

  double d12 = sqrt(d12sq);
  double d12inv = 1.0 / d12;
  double d12inv3 = d12inv * d12inv * d12inv;
  double exponential = exp(-d12 * b); // b = 1/rho
  double erfc_r = erfc(RI_ALPHA * d12) * d12inv;
  p2 = a * exponential - c * d12inv3 * d12inv3;
  p2 += qq * (erfc_r - ri.v_rc - ri.dv_rc * (d12 - ri.cutoff));
  f2 = 6.0 * c * (d12inv3 * d12inv3 * d12inv) - a * exponential * b;
  f2 -= qq * (erfc_r * d12inv + RI_PI_FACTOR * d12inv * exp(-RI_ALPHA_SQ * d12sq) + ri.dv_rc);
  f2 *= d12inv;
}

// force evaluation kernel
static __global__ void gpu_find_force(
  RI_Para ri,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const int shift,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_potential)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  double s_fx = 0.0;                                   // force_x
  double s_fy = 0.0;                                   // force_y
  double s_fz = 0.0;                                   // force_z
  double s_pe = 0.0;                                   // potential energy
  double s_sxx = 0.0;                                  // virial_stress_xx
  double s_sxy = 0.0;                                  // virial_stress_xy
  double s_sxz = 0.0;                                  // virial_stress_xz
  double s_syx = 0.0;                                  // virial_stress_yx
  double s_syy = 0.0;                                  // virial_stress_yy
  double s_syz = 0.0;                                  // virial_stress_yz
  double s_szx = 0.0;                                  // virial_stress_zx
  double s_szy = 0.0;                                  // virial_stress_zy
  double s_szz = 0.0;                                  // virial_stress_zz

  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1] - shift;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2] - shift;

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12sq = x12 * x12 + y12 * y12 + z12 * z12;

      double p2, f2;

      // RI
      if (d12sq >= ri.cutoff * ri.cutoff) {
        continue;
      }
      find_p2_and_f2(type1, type2, ri, d12sq, p2, f2);

      // treat two-body potential in the same way as many-body potential
      double f12x = f2 * x12 * 0.5;
      double f12y = f2 * y12 * 0.5;
      double f12z = f2 * z12 * 0.5;
      double f21x = -f12x;
      double f21y = -f12y;
      double f21z = -f12z;

      // accumulate force
      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      // accumulate potential energy and virial
      s_pe += p2 * 0.5; // two-body potential
      s_sxx += x12 * f21x;
      s_sxy += x12 * f21y;
      s_sxz += x12 * f21z;
      s_syx += y12 * f21x;
      s_syy += y12 * f21y;
      s_syz += y12 * f21z;
      s_szx += z12 * f21x;
      s_szy += z12 * f21y;
      s_szz += z12 * f21z;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;
  }
}

// Find force and related quantities for pair potentials (A wrapper)
void RI::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

  gpu_find_force<<<grid_size, BLOCK_SIZE_FORCE>>>(
    ri_para, number_of_atoms, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    type.data(), type_shift, position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2, force_per_atom.data(),
    force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
    virial_per_atom.data(), potential_per_atom.data());
  CUDA_CHECK_KERNEL
}
